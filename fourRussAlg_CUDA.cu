

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <stdbool.h>
#include <sys/time.h>

struct timeval t0;
struct timeval tic();
float toc(struct timeval begin);
int** one2twodim(int* A, int len);
int*  two2onedim(int** A, int row, int col);
void  testBMM(int **A, int **B, int **D, int size);
int   bin2dec(int *binvec, int len);
int** boolMatrix(int row, int col);
int** matrixOR(int **mat1, int **mat2, int size);
int** matrixAND(int **mat1, int **mat2, int size);
int   vectOR(int v1, int v2);
void  vectCpy(int *v1, int *v2, int start, int end);
int*  RowFromBottom(int **B, int partB, int t, int pos, int n);
void  printArray(int **A, int *B, int len, int dim);
int** calc_rowSums(int **B, int n, int t, int partB);
int* BoolMatrixMult(int *A, int **B, int n, int t, int **Filt, bool filtered);


__device__ void vectOR2(int *v1, int *v2, int n, int *v3, bool newMat){
    
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    if( newMat ){
            v3[id] = ( v1[id] | v2[id] );
    }
    else{
        for(int j = 0; j < n; j++)
            v1[id*n+j] = ( v1[id*n+j] | v2[id*n+j] );
    }
    
}


__global__ void fourRus(int *A, int *RS, int n, int t, int partA, int *Cpar, int *dC){

    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int bin, dec = 0;

    for(int k=0, k2=t-1; k < t; k++, k2--){
        bin =  A[n*id+t*partA+k2];
        dec += bin * pow(2, k);
    }

    for(int j = 0; j < n; j++)
        Cpar[n*id+j] = RS[n*dec+j];
    
    vectOR2(dC, Cpar, n, NULL, false);
}


__global__ void help(int *v1, int *v2, int n, int *v3, bool newMat){
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    v3[id] = ( v1[id] | v2[id] );
}





int main(void){

    int n = 120,
        t = 3;
    
    int **A1 = boolMatrix(n, n);
    int *A = two2onedim(A1, n, n);
    int **B = boolMatrix(n, n);

    int *C = BoolMatrixMult(A, B, n, t, NULL, false);
    int **C2=one2twodim(C, n);
    testBMM(A1, B, C2, n);
    
    free(A);
    free(A1);
    free(B);
    free(C);
    free(C2);
}


int* BoolMatrixMult(int *A, int **B, int n, int t, int **Filt, bool filtered){
  
    int **Cp  = (int **)malloc(n * sizeof(int *));
    int *C   = (int *)malloc(n * n * sizeof(int));
    int *help = (int  *)malloc(n * n * sizeof(int));
    int *RS1 = (int  *)malloc(n * pow(2,t) * sizeof(int));
    for(int g = 0; g < n; g++) {
        Cp[g] = (int *)malloc(n * sizeof(int));
    }

    
    int **RS  = (int **)malloc(pow(2,t) * sizeof(int *));
    for(int g = 0; g < pow(2,t); g++) RS[g]=(int *)malloc(n * sizeof(int));

    int *Cpar, *dA, *dRS, size  = n*n*sizeof(int), *dC,
                          size2 = n*pow(2, t)*sizeof(int);

    hipMalloc((void **)&dA  , size);
    hipMalloc((void **)&dRS , size2);
    hipMalloc((void **)&Cpar, size);
    hipMalloc((void **)&dC  , size);

    hipMemcpy(dA, A, size, hipMemcpyHostToDevice);
    t0 = tic();

    for(int i = 0; i < n/t; i++){

        RS = calc_rowSums(B, n, t, i);   
        RS1 = two2onedim(RS, pow(2, t), n);     
        hipMemcpy(dRS, RS1, size2, hipMemcpyHostToDevice);

        fourRus<<<20, n/20>>>(dA, dRS, n, t, i, Cpar, dC);
        hipDeviceSynchronize();
      
    }
    
    hipMemcpy(C, dC, size, hipMemcpyDeviceToHost);
    float dur = toc(t0);
    printf("~ Duration: %f\n", dur);
    
    hipFree(dA);
    hipFree(dRS);
    hipFree(Cpar);
    hipFree(dC);
    return C;
}



/* All possible row sums of matrix */
int** calc_rowSums(int **B, int n, int t, int partB){

    int indx, bp, k;
    int **RS  = (int **)calloc(pow(2, t) , sizeof(int *));
    int *temp = (int  *)calloc(n , sizeof(int  ));
    for(int g = 0; g < pow(2, t); g++) RS[g] = (int *)calloc(n , sizeof(int));

    bp = 1;
    k  = 0;

    int *dRS, *dtemp, *dres, size = n*sizeof(int);
    hipMalloc((void **)&dRS, size);
    hipMalloc((void **)&dtemp, size);
    hipMalloc((void **)&dres, size);
    
    for(int j = 1; j < pow(2, t); j++){
        
        indx  = j - pow(2, k); 
        temp  = RowFromBottom(B, partB, t, k+1, n);

        hipMemcpy(dRS, RS[indx], size, hipMemcpyHostToDevice);
        hipMemcpy(dtemp, temp, size, hipMemcpyHostToDevice);

        help<<<20, n/20>>>(dRS, dtemp, n, dres, true);
        hipMemcpy(RS[j], dres, size, hipMemcpyDeviceToHost);
        
        if(bp == 1){
            bp = j + 1;
            k ++;
        }
        else bp --;
    }

    return RS;
}


/* Accesses array from bottom */
int* RowFromBottom(int **B, int partB, int t, int pos, int n){
    int *res=(int *)malloc(n * sizeof(int));
    for(int i=0; i<n; i++) res[i]=B[t*partB+t-pos][i];
    return res;
}


/* copies vector */
void vectCpy(int *v1, int *v2, int start, int end){

    for(int i = start; i < end; i++)
        v1[i] = v2[i];
}


/* ORs two vectors */
int vectOR(int v1, int v2){
    
    int v3 = ( v1 | v2 );
    return v3;
}


/* ORs 2-D matrices */
int** matrixOR(int **mat1, int **mat2, int size){

    for(int i = 0; i < size; i++)
        for(int j = 0; j < size; j++)
            mat1[i][j] = ( mat1[i][j] | mat2[i][j] );

    return mat1;
}


/* ANDs 2-D matrices */
int** matrixAND(int **mat1, int **mat2, int size){

    for(int i = 0; i < size; i++)
        for(int j = 0; j < size; j++)
            mat1[i][j] = ( mat1[i][j] & mat2[i][j] );

    return mat1;
}


/* Binary vector to int */
int bin2dec(int *binvec, int len){

    int decval = 0;

    for(int i=0, i2=len-1; i < len; i++, i2--)
        decval += binvec[i2] * pow(2, i);
    
    return decval;
}


/* Random boolean matrix */
int** boolMatrix(int row, int col){

    srand(time( NULL ));
    int** mat = (int **)malloc(row * sizeof(int *));
    for(int i=0; i<row; i++) mat[i] = (int *)malloc(col * sizeof(int));

    for(int i=0; i<row; i++)
        for(int j=0; j<col; j++)
            mat[i][j] = rand() % 2;

    return mat;
}


int** one2twodim(int* A, int len){

    int** D = (int **)calloc(len , sizeof(int *));
    for(int f = 0; f < len; f++) D[f] = (int *)calloc(len , sizeof(int));
    int cnt = 0;

    for(int i = 0; i < len; i++)
        for(int j = 0; j < len; j++)
            D[i][j] = A[cnt++];

    return D;
}


/* 2D to 1D */
int* two2onedim(int** A, int row, int col){

    int* C = (int *)calloc(row * col , sizeof(int));
    int cnt = 0;

    for(int i = 0; i < row; i++)
        for(int j = 0; j < col; j++)
            C[cnt++] = A[i][j];

    return C;
}

void printArray(int **A, int *B, int len, int dim){

    if(dim == 2){
        for(int i = 0; i < len; i++){
            for(int j = 0; j < len; j++){
                printf("%d ", A[i][j]);
            }
            printf("\n");
        }
    }
    else{
        for(int i = 0; i < len; i++) printf("%d ", B[i]);
    }
    printf("\n");
}


/* BMM tester */
void testBMM(int **A, int **B, int **D, int size){

    int **C = (int **)calloc(size , sizeof(int *));
    for(int i=0; i<size; i++) C[i] = (int *)calloc(size , sizeof(int));

    for(int r=0; r<size; r++)
        for(int i=0; i< size; i++)
            for(int j=0; j<size; j++)
                C[r][i] = C[r][i] | ( A[r][j] & B[j][i] );
            
    for(int i=0; i< size; i++)
        for(int j=0; j<size; j++)
            if( D[i][j] != C[i][j] )
            {
                printf("ERROR!\nIncorrect element calculation!\n");
                exit(0);
            }

    printf("CORRECT!\n");
}


struct timeval tic(){
    
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv;
}


float toc(struct timeval begin){
    
    struct timeval end;
    gettimeofday(&end, NULL);
    float stime = ((double)(end.tv_sec-begin.tv_sec)*1000)+
                            ((double)(end.tv_usec-begin.tv_usec)/1000);
    stime /= 1000;
    return stime;
}